#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <cmath>

#include "caffe/layers/label_specific_affine_layer.hpp"

namespace caffe {

  template <typename Dtype>
  __global__ void LabelSpecificAffineForward(const int n, const int dim, const Dtype* bottom_data, const Dtype* label,
                                                 Dtype* top_data, Dtype scale, Dtype bias) {
    CUDA_KERNEL_LOOP(index, n) {
      int gt = static_cast<int>(label[index]);
      top_data[index * dim + gt] = bottom_data[index * dim + gt] * scale + bias;
    }
  }

  template <typename Dtype>
  __global__ void LabelSpecificAffineBackward(const int n, const int dim, const Dtype* top_diff, const Dtype* label,
                                              Dtype* bottom_diff, Dtype scale) {
    CUDA_KERNEL_LOOP(index, n) {
      int gt = static_cast<int>(label[index]);
      bottom_diff[index * dim + gt] = top_diff[index * dim + gt] * scale;
    }
  }

  template <typename Dtype>
  __global__ void LabelSpecificAffineBackwardScale(const int n, const int dim, const Dtype* top_diff, const Dtype* label,
                                                   const Dtype* bottom_data, Dtype* selected_value) {
    CUDA_KERNEL_LOOP(index, n) {
      int gt = static_cast<int>(label[index]);
      selected_value[index] = top_diff[index * dim + gt] * bottom_data[index * dim + gt];
    }
  }

  template <typename Dtype>
  __global__ void LabelSpecificAffineBackwardBias(const int n, const int dim, const Dtype* top_diff, const Dtype* label,
                                                  const Dtype* bottom_data, Dtype* selected_value) {
    CUDA_KERNEL_LOOP(index, n) {
      int gt = static_cast<int>(label[index]);
      selected_value[index] = top_diff[index * dim + gt];
    }
  }

  template <typename Dtype>
  void LabelSpecificAffineLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                                    const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* label_data = bottom[1]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    const Dtype* scale_bias = (bottom.size() == 3) ? bottom[2]->cpu_data() : this->blobs_[0]->cpu_data();

    int num = bottom[0]->num();
    int count = bottom[0]->count();
    int dim = count / num;

    if (this->phase_ == TEST) {
      scale = Dtype(1);
      bias = Dtype(0);
    }
    else {
      if (auto_tune_) {
        scale = scale_bias[0];
        bias = scale_bias[1];
      }
      else {
        scale = scale_base_ * pow(((Dtype)1. + scale_gamma_ * iteration_), scale_power_);
        bias = bias_base_ + pow(((Dtype)1. + bias_gamma_ * iteration_), bias_power_) - (Dtype)1.;
        scale = std::min(scale, scale_max_);
        bias = std::min(bias, bias_max_);
        iteration_++;
      }
    }

    caffe_copy(count, bottom_data, top_data);
    if (!transform_test_ && this->phase_ == TEST) return;

    // NOLINT_NEXT_LINE(whitespace/operators)
    LabelSpecificAffineForward<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
      num, dim, bottom_data, label_data, top_data, scale, bias);
    CUDA_POST_KERNEL_CHECK;
  }

  template <typename Dtype>
  void LabelSpecificAffineLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                                     const vector<bool>& propagate_down,
                                                     const vector<Blob<Dtype>*>& bottom) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* label_data = bottom[1]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* top_data = top[0]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    Dtype* scale_bias_diff = bottom.size() == 3 ? bottom[2]->mutable_cpu_diff() : this->blobs_[0]->mutable_cpu_diff();

    int num = bottom[0]->num();
    int count = bottom[0]->count();
    int dim = count / num;
    if (propagate_down[0]) {
      caffe_copy(count, top_diff, bottom_diff);
      if (!transform_test_ && this->phase_ == TEST) return;

      // NOLINT_NEXT_LINE(whitespace/operators)
      LabelSpecificAffineBackward<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
        num, dim, top_diff, label_data, bottom_diff, scale);
      CUDA_POST_KERNEL_CHECK;
    }

    if (auto_tune_ || (bottom.size() == 3 && propagate_down[2]) {
      // NOLINT_NEXT_LINE(whitespace/operators)
      LabelSpecificAffineBackwardScale<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
        num, dim, top_diff, label_data, bottom_data, selected_value_.mutable_gpu_data());
      caffe_gpu_dot(num, selected_value_.gpu_data(), sum_multiplier_.gpu_data(), scale_bias_diff);

      // NOLINT_NEXT_LINE(whitespace/operators)
      LabelSpecificAffineBackwardBias<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
        num, dim, top_diff, label_data, bottom_data, selected_value_.mutable_gpu_data());
      caffe_gpu_dot(num, selected_value_.gpu_data(), sum_multiplier_.gpu_data(), scale_bias_diff+1);
    }
  }


  INSTANTIATE_LAYER_GPU_FUNCS(LabelSpecificAffineLayer);


}  // namespace caffe
