#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <cmath>

#include "caffe/layers/label_specific_margin_layer.hpp"

namespace caffe {

  template <typename Dtype>
  __global__ void ArccosForward(const int n, const Dtype* in, Dtype* out) {
    CUDA_KERNEL_LOOP(index, n) {
      out[index] = Dtype(acos(in[index]) / M_PI * 180.0);
    }
  }

  template <typename Dtype>
  __global__ void CreateMask(const int num, const int dim, const Dtype* label, Dtype* positive_mask, Dtype* negative_mask) {
    CUDA_KERNEL_LOOP(index, num) {
      int gt = static_cast<int>(label[index]);
      positive_mask[index*dim + gt] = Dtype(1);
      negative_mask[index*dim + gt] = Dtype(0);
    }
  }

  template <typename Dtype>
  __global__ void LabelSpecificMarginForward(const int n, const int dim, const Dtype* bottom_data, const Dtype* label,
                                             Dtype* top_data, Dtype margin) {
    CUDA_KERNEL_LOOP(index, n) {
      int l = static_cast<int>(label[index]);
      top_data[index * dim + l] = bottom_data[index * dim + l] * cosf(margin / 180 * M_PI) -
        sqrt(1 - bottom_data[index * dim + l] * bottom_data[index * dim + l] + 1e-12) * sinf(margin / 180 * M_PI);
    }
  }

  template <typename Dtype>
  __global__ void LabelSpecificMarginBackward(const int n, const int dim, const Dtype* top_diff, const Dtype* label,
                                                     Dtype* bottom_diff, const Dtype* bottom_data, Dtype margin) {
    CUDA_KERNEL_LOOP(index, n) {
      int l = static_cast<int>(label[index]);
      bottom_diff[index * dim + l] = top_diff[index * dim + l] * (cosf(margin / 180 * M_PI) -
                                                    bottom_data[index * dim + l] / sqrt(1 - bottom_data[index * dim + l] * bottom_data[index * dim + l] + 1e-12) * sinf(margin / 180 * M_PI));
    }
  }

template <typename Dtype>
void LabelSpecificMarginLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                                  const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* label_data = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* margin = this->blobs_[0]->mutable_cpu_data();

  int num = bottom[0]->num();
  int count = bottom[0]->count();
  int dim = count / num;

  if (has_margin_base_) {
    margin[0] = margin_base_ + pow(((Dtype)1. + gamma_ * iter_), power_) - 1;
    iter_++;
  }
  if (has_margin_max_) {
    margin[0] = std::min(margin[0], margin_max_);
  }

  if (top.size() >= 2 && auto_tune_) {
    Dtype *positive_mask_data = positive_mask.mutable_gpu_data();
    Dtype *negative_mask_data = negative_mask.mutable_gpu_data();
    caffe_gpu_set(count, Dtype(0), positive_mask_data);
    caffe_gpu_set(count, Dtype(1), negative_mask_data);
    // NOLINT_NEXT_LINE(whitespace/operators)
    CreateMask<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
      num, dim, label_data, positive_mask.mutable_gpu_data(), negative_mask.mutable_gpu_data());
    CUDA_POST_KERNEL_CHECK;

    Dtype positive_mean;
    Dtype positive_std;
    Dtype negative_mean;
    Dtype negative_std;
    
    // NOLINT_NEXT_LINE(whitespace/operators)
    ArccosForward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> > (
      count, bottom_data, bottom_angle.mutable_gpu_data());
    caffe_gpu_powx(count, bottom_angle.gpu_data(), Dtype(2), bottom_square.mutable_gpu_data());
    caffe_gpu_dot(count, bottom_angle.gpu_data(), positive_mask.gpu_data(), &positive_mean);
    caffe_gpu_dot(count, bottom_square.gpu_data(), positive_mask.gpu_data(), &positive_std);
    caffe_gpu_dot(count, bottom_angle.gpu_data(), negative_mask.gpu_data(), &negative_mean);
    caffe_gpu_dot(count, bottom_square.gpu_data(), negative_mask.gpu_data(), &negative_std);

    positive_mean /= num;
    positive_std = sqrt(positive_std / num - positive_mean * positive_mean);
    negative_mean /= num * (dim - 1);
    negative_std = sqrt(negative_std / num / (dim - 1) - negative_mean * negative_mean);
    
    if (this->phase_ == TEST) {
      top[1]->mutable_cpu_data()[0] = margin[0];
      top[1]->mutable_cpu_data()[1] = positive_mean;
      top[1]->mutable_cpu_data()[2] = positive_std;
      top[1]->mutable_cpu_data()[3] = negative_mean;
      top[1]->mutable_cpu_data()[4] = negative_std;
    }
    else {
      if (iter_ == 1) {
        margin[1] = positive_mean;
        margin[2] = positive_std;
        margin[3] = negative_mean;
        margin[4] = negative_std;
      }
      else {
        margin[1] = 0.99 * margin[1] + 0.01 * positive_mean;
        margin[2] = 0.99 * margin[2] + 0.01 * positive_std;
        margin[3] = 0.99 * margin[3] + 0.01 * negative_mean;
        margin[4] = 0.99 * margin[4] + 0.01 * negative_std;
      }
     
      margin[0] = (margin[3] - margin[1]) / (margin[2] + margin[3]) * margin[2];
      caffe_copy(5, this->blobs_[0]->cpu_data(), top[1]->mutable_cpu_data());
    }
  }
  if (top.size() >= 2 && !auto_tune_) {
    top[1]->mutable_cpu_data()[0] = margin[0];
  }

  caffe_copy(count, bottom_data, top_data);
  if (!margin_on_test_ && this->phase_ == TEST) return;

  if (margin[0] != Dtype(0.0)) {
    // NOLINT_NEXT_LINE(whitespace/operators)
    LabelSpecificMarginForward<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
      num, dim, bottom_data, label_data, top_data, margin[0]);
    CUDA_POST_KERNEL_CHECK;
  }
}

template <typename Dtype>
void LabelSpecificMarginLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* label_data = bottom[1]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    Dtype* margin = this->blobs_[0]->mutable_cpu_data();

    int num = bottom[0]->num();
    int count = bottom[0]->count();
    int dim = count / num;

    caffe_copy(count, top_diff, bottom_diff);
    if (!margin_on_test_ && this->phase_ == TEST) return;
    
    if (margin[0] != Dtype(0.0)) {
      // NOLINT_NEXT_LINE(whitespace/operators)
      LabelSpecificMarginBackward<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
        num, dim, top_diff, label_data, bottom_diff, bottom_data, margin[0]);
      CUDA_POST_KERNEL_CHECK;
    }
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(LabelSpecificMarginLayer);


}  // namespace caffe
